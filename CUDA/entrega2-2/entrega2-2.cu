
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
Programa en Cuda que realiza la Multiplicación de Matrices MxN (4x4),
utilizando un bloque de hilos y memoria global
*/

const int dim = 4;

__global__ void multiplicarMatriz(int (*M_d)[dim], int (*N_d)[dim],
                                  int (*R_d)[dim])
{
    int valor = 0;
    for (int k = 0; k < dim; k++)
    {
        int Melemento = M_d[threadIdx.y][k];
        int Nelemento = N_d[k][threadIdx.x];
        
        valor += Melemento * Nelemento;
    }
    
    R_d[threadIdx.y][threadIdx.x] = valor;
}

int main(int argc, char **argv)
{
    // Declarar todas las variables
    const int longitud = dim * dim * sizeof(int);
    int M_h[dim][dim] = 
    {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16}
    };
    int N_h[dim][dim] = 
    {
        {4, 3, 2, 1},
        {8, 7, 6, 5},
        {12, 11, 10, 9},
        {16, 15, 14, 13}
    };
    int R_h[dim][dim] = {};
    int (*M_d)[dim];
    int (*N_d)[dim];
    int (*R_d)[dim];
    
    // Asignar memoria en el dispositivo
    hipMalloc((void **) &M_d, longitud);
    hipMalloc((void **) &N_d, longitud);
    hipMalloc((void **) &R_d, longitud);
    
    // Transferir datos al dispositivo
    hipMemcpy(M_d, M_h, longitud, hipMemcpyHostToDevice);
    hipMemcpy(N_d, N_h, longitud, hipMemcpyHostToDevice);
    
    // Ejecutar kernel en el dispositivo, dos bloques con ochos hilos cada uno
    dim3 bloques(1, 1);
    dim3 hilos(dim, dim);
    multiplicarMatriz<<<bloques, hilos>>>(M_d, N_d, R_d);
    
    // Transferir resultados al anfitrión
    hipMemcpy(R_h, R_d, longitud, hipMemcpyDeviceToHost);
    
    // Mostrar resultados
    printf("{\n"
           "  {1, 2, 3, 4},\n"
           "  {5, 6, 7, 8},\n"
           "  {9, 10, 11, 12},\n"
           "  {13, 14, 15, 16},\n"
           "}\n"
           "*\n"
           "{\n"
           "  {4, 3, 2, 1},\n"
           "  {8, 7, 6, 5},\n"
           "  {12, 11, 10, 9},\n"
           "  {16, 15, 14, 13},\n"
           "}\n"
           "=\n"
           "{\n"
           "  {%d, %d, %d, %d},\n"
           "  {%d, %d, %d, %d},\n"
           "  {%d, %d, %d, %d},\n"
           "  {%d, %d, %d, %d},\n"
           "}\n",
           R_h[0][0], R_h[0][1], R_h[0][2], R_h[0][3],
           R_h[1][0], R_h[1][1], R_h[1][2], R_h[1][3],
           R_h[2][0], R_h[2][1], R_h[2][2], R_h[2][3],
           R_h[3][0], R_h[3][1], R_h[3][2], R_h[3][3]);
    
    // Liberar memoria del dispositivo
    hipFree(M_d);
    hipFree(N_d);
    hipFree(R_d);
    
    return 0;
}

